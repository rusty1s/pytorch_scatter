#include "hip/hip_runtime.h"
#include "scatter_cuda.h"

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>

#include "reducer.cuh"
#include "utils.cuh"

#define THREADS 256
#define BLOCKS(N) (N + THREADS - 1) / THREADS

template <typename scalar_t, ReductionType REDUCE>
__global__ void
scatter_kernel(const scalar_t *src_data,
               const at::cuda::detail::TensorInfo<int64_t, int> index_info,
               scalar_t *out_data, int E, int K, int N, int numel) {

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  int b = thread_idx / (E * K);
  int k = thread_idx % K;

  if (thread_idx < numel) {
    int offset = at::cuda::detail::IndexToOffset<int64_t, int, -1>::get(
        thread_idx, index_info);
    int64_t idx = index_info.data[offset];

    Reducer<scalar_t, REDUCE>::atomic_write(out_data + b * N * K + idx * K + k,
                                            src_data[thread_idx]);
  }
}

template <typename scalar_t>
__global__ void
scatter_arg_kernel(const scalar_t *src_data,
                   const at::cuda::detail::TensorInfo<int64_t, int> index_info,
                   const scalar_t *out_data, int64_t *arg_out_data, int E,
                   int K, int N, int numel) {

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;

  int b = thread_idx / (E * K);
  int e = (thread_idx / K) % E;
  int k = thread_idx % K;

  if (thread_idx < numel) {
    int offset = at::cuda::detail::IndexToOffset<int64_t, int, -1>::get(
        thread_idx, index_info);
    int64_t idx = index_info.data[offset];

    if (src_data[thread_idx] == out_data[b * N * K + idx * K + k]) {
      arg_out_data[b * N * K + idx * K + k] = e;
    }
  }
}

std::tuple<torch::Tensor, torch::optional<torch::Tensor>>
scatter_cuda(torch::Tensor src, torch::Tensor index, int64_t dim,
             torch::optional<torch::Tensor> optional_out,
             torch::optional<int64_t> dim_size, std::string reduce) {
  CHECK_CUDA(src);
  CHECK_CUDA(index);
  if (optional_out.has_value())
    CHECK_CUDA(optional_out.value());
  hipSetDevice(src.get_device());

  CHECK_INPUT(src.dim() == index.dim());
  for (auto i = 0; i < index.dim() - 1; i++)
    CHECK_INPUT(src.size(i) >= index.size(i));

  src = src.contiguous();

  torch::Tensor out;
  if (optional_out.has_value()) {
    out = optional_out.value().contiguous();
    for (auto i = 0; i < out.dim(); i++)
      if (i != dim)
        CHECK_INPUT(src.size(i) == out.size(i));
  } else {
    auto sizes = src.sizes().vec();
    if (dim_size.has_value())
      sizes[dim] = dim_size.value();
    else if (index.numel() == 0)
      sizes[dim] = 0;
    else {
      sizes[dim] = 1 + index.max().cpu().data_ptr<int64_t>()[0];
    }
    out = torch::empty(sizes, src.options());
  }

  torch::optional<torch::Tensor> arg_out = torch::nullopt;
  int64_t *arg_out_data = nullptr;
  if (reduce2REDUCE.at(reduce) == MIN || reduce2REDUCE.at(reduce) == MAX) {
    arg_out = torch::full_like(out, src.size(dim), index.options());
    arg_out_data = arg_out.value().data_ptr<int64_t>();
  }

  if (src.numel() == 0) {
    if (!optional_out.has_value())
      out.fill_(0);
    return std::make_tuple(out, arg_out);
  }

  auto B = 1;
  for (auto i = 0; i < dim; i++)
    B *= src.size(i);
  auto E = src.size(dim);
  auto K = src.numel() / (B * E);
  auto N = out.size(dim);

  auto index_info = at::cuda::detail::getTensorInfo<int64_t, int>(index);
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES_AND2(at::ScalarType::Half, at::ScalarType::BFloat16, src.scalar_type(), "_", [&] {
    auto src_data = src.data_ptr<scalar_t>();
    auto out_data = out.data_ptr<scalar_t>();

    AT_DISPATCH_REDUCTION_TYPES(reduce, [&] {
      if (!optional_out.has_value())
        out.fill_(Reducer<scalar_t, REDUCE>::init());

      scatter_kernel<scalar_t, REDUCE>
          <<<BLOCKS(src.numel()), THREADS, 0, stream>>>(
              src_data, index_info, out_data, E, K, N, src.numel());

      if (!optional_out.has_value() && (REDUCE == MIN || REDUCE == MAX))
        out.masked_fill_(out == Reducer<scalar_t, REDUCE>::init(), (scalar_t)0);

      if (REDUCE == MIN || REDUCE == MAX)
        scatter_arg_kernel<scalar_t>
            <<<BLOCKS(src.numel()), THREADS, 0, stream>>>(
                src_data, index_info, out_data, arg_out_data, E, K, N,
                src.numel());
    });
  });

  return std::make_tuple(out, arg_out);
}

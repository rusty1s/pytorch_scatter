#include "hip/hip_runtime.h"
#include "segment_csr_cuda.h"

#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>

#include "index_info.cuh"
#include "reducer.cuh"
#include "utils.cuh"

#define THREADS 256
#define BLOCKS(TB, N) (TB * N + THREADS - 1) / THREADS
#define FULL_MASK 0xffffffff

template <typename scalar_t, ReductionType REDUCE, int TB>
__global__ void
segment_csr_kernel(const scalar_t *src_data,
                   const at::cuda::detail::TensorInfo<int64_t, int> indptr_info,
                   scalar_t *out_data, int64_t *arg_out_data, size_t N,
                   size_t E) {

  // Each warp processes exactly `32/TB` rows and aggregates all row values
  // via a parallel reduction.

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row_idx = thread_idx / TB;
  int lane_idx = thread_idx & (TB - 1);

  using cuda_scalar_t =
      typename std::conditional<std::is_same<scalar_t, at::Half>::value, __half,
                                scalar_t>::type;

  if (row_idx < N) {
    int offset = IndexPtrToOffset<int64_t>::get(row_idx, indptr_info);
    int64_t row_start = __ldg(indptr_info.data + offset);
    int64_t row_end = __ldg(indptr_info.data + offset +
                            indptr_info.strides[indptr_info.dims - 1]);

    scalar_t val = Reducer<scalar_t, REDUCE>::init();
    int64_t arg, arg_tmp;

    offset = (row_idx / (indptr_info.sizes[indptr_info.dims - 1] - 1)) * E;
    for (int64_t src_idx = row_start + lane_idx; src_idx < row_end;
         src_idx += TB) {
      Reducer<scalar_t, REDUCE>::update(&val, src_data[offset + src_idx], &arg,
                                        src_idx);
    }

#pragma unroll
    for (int i = TB / 2; i > 0; i /= 2) {
      // Parallel reduction inside a single warp.
      if (REDUCE == MIN || REDUCE == MAX)
        arg_tmp = __shfl_down_sync(FULL_MASK, arg, i);
      Reducer<scalar_t, REDUCE>::update(
          &val, __shfl_down_sync(FULL_MASK, (cuda_scalar_t)val, i), &arg,
          arg_tmp);
    }

    if (lane_idx == 0) {
      Reducer<scalar_t, REDUCE>::write(out_data + row_idx, val,
                                       arg_out_data + row_idx, arg,
                                       row_end - row_start);
    }
  }
}

template <typename scalar_t, ReductionType REDUCE>
__global__ void segment_csr_broadcast_kernel(
    const scalar_t *src_data,
    const at::cuda::detail::TensorInfo<int64_t, int> indptr_info,
    scalar_t *out_data, int64_t *arg_out_data, size_t N, size_t K, size_t E) {

  // Each thread processes exactly one row. It turned out that is more
  // efficient than using shared memory due to avoiding synchronization
  // barriers.

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row_idx = thread_idx / K;
  int lane_idx = thread_idx % K;

  if (thread_idx < N * K) {
    int offset = IndexPtrToOffset<int64_t>::get(row_idx, indptr_info);
    int64_t row_start = __ldg(indptr_info.data + offset);
    int64_t row_end = __ldg(indptr_info.data + offset +
                            indptr_info.strides[indptr_info.dims - 1]);

    scalar_t val = Reducer<scalar_t, REDUCE>::init();
    int64_t arg;

    offset = (row_idx / (indptr_info.sizes[indptr_info.dims - 1] - 1)) * E * K;
    for (int64_t src_idx = row_start; src_idx < row_end; src_idx++) {
      Reducer<scalar_t, REDUCE>::update(
          &val, src_data[offset + K * src_idx + lane_idx], &arg, src_idx);
    }

    Reducer<scalar_t, REDUCE>::write(out_data + thread_idx, val,
                                     arg_out_data + thread_idx, arg,
                                     row_end - row_start);
  }
}

std::tuple<torch::Tensor, torch::optional<torch::Tensor>>
segment_csr_cuda(torch::Tensor src, torch::Tensor indptr,
                 torch::optional<torch::Tensor> optional_out,
                 std::string reduce) {
  CHECK_CUDA(src);
  CHECK_CUDA(indptr);
  if (optional_out.has_value())
    CHECK_CUDA(optional_out.value());
  hipSetDevice(src.get_device());

  CHECK_INPUT(src.dim() >= indptr.dim());

  auto sizes = indptr.sizes().vec();
  for (auto i = 0; i < indptr.dim() - 1; i++)
    sizes[i] = src.size(i);
  indptr = indptr.expand(sizes);

  auto dim = indptr.dim() - 1;

  src = src.contiguous();

  torch::Tensor out;
  if (optional_out.has_value()) {
    out = optional_out.value().contiguous();
    for (int i = 0; i < out.dim(); i++)
      if (i != dim)
        CHECK_INPUT(src.size(i) == out.size(i));
    CHECK_INPUT(src.numel() == 0 || out.size(dim) == indptr.size(dim) - 1);
  } else {
    sizes = src.sizes().vec();
    sizes[dim] = std::max<int64_t>(indptr.size(dim) - 1, 0);
    out = torch::empty(sizes, src.options());
  }

  torch::optional<torch::Tensor> arg_out = torch::nullopt;
  int64_t *arg_out_data = nullptr;
  if (reduce2REDUCE.at(reduce) == MIN || reduce2REDUCE.at(reduce) == MAX) {
    arg_out = torch::full(out.sizes(), src.size(dim), indptr.options());
    arg_out_data = arg_out.value().data_ptr<int64_t>();
  }

  if (src.numel() == 0) {
    if (!optional_out.has_value())
      out.fill_(0);
    return std::make_tuple(out, arg_out);
  }

  auto N = out.size(dim) * (indptr.numel() / indptr.size(-1));
  auto K = out.numel() / N;
  auto E = src.size(dim);

  auto indptr_info = at::cuda::detail::getTensorInfo<int64_t, int>(indptr);
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, src.scalar_type(), "_", [&] {
    auto src_data = src.data_ptr<scalar_t>();
    auto out_data = out.data_ptr<scalar_t>();

    AT_DISPATCH_REDUCTION_TYPES(reduce, [&] {
      if (K == 1) {
        segment_csr_kernel<scalar_t, REDUCE, 1>
            <<<BLOCKS(32, N), THREADS, 0, stream>>>(
                src_data, indptr_info, out_data, arg_out_data, N, E);
      } else {
        segment_csr_broadcast_kernel<scalar_t, REDUCE>
            <<<BLOCKS(1, N * K), THREADS, 0, stream>>>(
                src_data, indptr_info, out_data, arg_out_data, N, K, E);
      }
    });
  });

  return std::make_tuple(out, arg_out);
}

template <typename scalar_t, int TB>
__global__ void
gather_csr_kernel(const scalar_t *src_data,
                  const at::cuda::detail::TensorInfo<int64_t, int> indptr_info,
                  scalar_t *out_data, size_t N, size_t E) {

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row_idx = thread_idx / TB;
  int lane_idx = thread_idx % TB;

  if (row_idx < N) {
    int offset = IndexPtrToOffset<int64_t>::get(row_idx, indptr_info);
    int row_start = __ldg(indptr_info.data + offset);
    int row_end = __ldg(indptr_info.data + offset +
                        indptr_info.strides[indptr_info.dims - 1]);
    scalar_t val = __ldg(src_data + row_idx);

    offset = (row_idx / (indptr_info.sizes[indptr_info.dims - 1] - 1)) * E;
    for (int out_idx = row_start + lane_idx; out_idx < row_end; out_idx += TB) {
      out_data[offset + out_idx] = val; // "Mostly" coalesced.
    }
  }
}

template <typename scalar_t>
__global__ void gather_csr_broadcast_kernel(
    const scalar_t *src_data,
    const at::cuda::detail::TensorInfo<int64_t, int> indptr_info,
    scalar_t *out_data, size_t N, size_t K, size_t E) {

  int thread_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int row_idx = thread_idx / K;
  int lane_idx = thread_idx % K;

  if (thread_idx < N * K) {
    int offset = IndexPtrToOffset<int64_t>::get(row_idx, indptr_info);
    int row_start = __ldg(indptr_info.data + offset);
    int row_end = __ldg(indptr_info.data + offset +
                        indptr_info.strides[indptr_info.dims - 1]);

    scalar_t val = src_data[thread_idx]; // Coalesced.

    offset = (row_idx / (indptr_info.sizes[indptr_info.dims - 1] - 1)) * E * K;
    for (int out_idx = row_start; out_idx < row_end; out_idx++) {
      out_data[offset + K * out_idx + lane_idx] = val; // "Mostly" coalesced.
    }
  }
}

torch::Tensor gather_csr_cuda(torch::Tensor src, torch::Tensor indptr,
                              torch::optional<torch::Tensor> optional_out) {
  CHECK_CUDA(src);
  CHECK_CUDA(indptr);
  if (optional_out.has_value())
    CHECK_CUDA(optional_out.value());
  hipSetDevice(src.get_device());

  CHECK_INPUT(src.dim() >= indptr.dim());

  auto sizes = indptr.sizes().vec();
  for (auto i = 0; i < indptr.dim() - 1; i++)
    sizes[i] = src.size(i);
  indptr = indptr.expand(sizes);

  auto dim = indptr.dim() - 1;
  CHECK_INPUT(src.size(dim) == 0 || src.size(dim) == indptr.size(dim) - 1);

  src = src.contiguous();

  torch::Tensor out;
  if (optional_out.has_value()) {
    out = optional_out.value().contiguous();
    for (auto i = 0; i < out.dim(); i++)
      if (i != dim)
        CHECK_INPUT(src.size(i) == out.size(i));
  } else {
    auto sizes = src.sizes().vec();
    if (src.numel() > 0) {
      sizes[dim] = indptr.flatten()[-1].cpu().data_ptr<int64_t>()[0];
    } else {
      sizes[dim] = 0;
    }
    out = torch::empty(sizes, src.options());
  }

  if (src.numel() == 0) {
    if (!optional_out.has_value())
      out.fill_(0);
    return out;
  }

  auto N = src.size(dim) * (indptr.numel() / indptr.size(-1));
  auto K = src.numel() / N;
  auto E = out.size(dim);

  auto indptr_info = at::cuda::detail::getTensorInfo<int64_t, int>(indptr);
  auto stream = at::cuda::getCurrentCUDAStream();
  AT_DISPATCH_ALL_TYPES_AND(at::ScalarType::Half, src.scalar_type(), "_", [&] {
    auto src_data = src.data_ptr<scalar_t>();
    auto out_data = out.data_ptr<scalar_t>();

    if (K == 1)
      gather_csr_kernel<scalar_t, 4><<<BLOCKS(1, 4 * N), THREADS, 0, stream>>>(
          src_data, indptr_info, out_data, N, E);
    else
      gather_csr_broadcast_kernel<scalar_t>
          <<<BLOCKS(1, N * K), THREADS, 0, stream>>>(src_data, indptr_info,
                                                     out_data, N, K, E);
  });

  return out;
}
